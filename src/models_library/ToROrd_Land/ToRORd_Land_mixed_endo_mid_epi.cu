#include "hip/hip_runtime.h"
#include "ToRORd_Land_mixed_endo_mid_epi.h"
#include <stddef.h>
#include <stdint.h>

__global__ void kernel_set_model_initial_conditions(real *sv, int num_volumes, size_t pitch, bool use_adpt_dt, real min_dt) {
    int threadID = blockDim.x * blockIdx.x + threadIdx.x;

    if (threadID < num_volumes) {
        // Default initial conditions for ENDO cell (from original Matlab script)
        *((real * )((char *) sv + pitch * 0) + threadID) = -8.863699e+01;
        *((real * )((char *) sv + pitch * 1) + threadID) = 1.189734e+01;
        *((real * )((char *) sv + pitch * 2) + threadID) = 1.189766e+01;
        *((real * )((char *) sv + pitch * 3) + threadID) = 1.412345e+02;
        *((real * )((char *) sv + pitch * 4) + threadID) = 1.412344e+02;
        *((real * )((char *) sv + pitch * 5) + threadID) = 7.267473e-05;
        *((real * )((char *) sv + pitch * 6) + threadID) = 6.337870e-05;
        *((real * )((char *) sv + pitch * 7) + threadID) = 1.532653e+00;
        *((real * )((char *) sv + pitch * 8) + threadID) = 1.533946e+00;
        *((real * )((char *) sv + pitch * 9) + threadID) = 8.280078e-04;
        *((real * )((char *) sv + pitch * 10) + threadID) = 6.665272e-01;
        *((real * )((char *) sv + pitch * 11) + threadID) = 8.260208e-01;
        *((real * )((char *) sv + pitch * 12) + threadID) = 8.260560e-01;
        *((real * )((char *) sv + pitch * 13) + threadID) = 8.258509e-01;
        *((real * )((char *) sv + pitch * 14) + threadID) = 1.668686e-04;
        *((real * )((char *) sv + pitch * 15) + threadID) = 5.228306e-01;
        *((real * )((char *) sv + pitch * 16) + threadID) = 2.859696e-01;
        *((real * )((char *) sv + pitch * 17) + threadID) = 9.591370e-04;
        *((real * )((char *) sv + pitch * 18) + threadID) = 9.996012e-01;
        *((real * )((char *) sv + pitch * 19) + threadID) = 5.934016e-01;
        *((real * )((char *) sv + pitch * 20) + threadID) = 4.886961e-04;
        *((real * )((char *) sv + pitch * 21) + threadID) = 9.996011e-01;
        *((real * )((char *) sv + pitch * 22) + threadID) = 6.546687e-01;
        *((real * )((char *) sv + pitch * 23) + threadID) = 9.500075e-32;
        *((real * )((char *) sv + pitch * 24) + threadID) = 1.000000e+00;
        *((real * )((char *) sv + pitch * 25) + threadID) = 9.392580e-01;
        *((real * )((char *) sv + pitch * 26) + threadID) = 1.000000e+00;
        *((real * )((char *) sv + pitch * 27) + threadID) = 9.998984e-01;
        *((real * )((char *) sv + pitch * 28) + threadID) = 9.999783e-01;
        *((real * )((char *) sv + pitch * 29) + threadID) = 4.448162e-04;
        *((real * )((char *) sv + pitch * 30) + threadID) = 7.550725e-04;
        *((real * )((char *) sv + pitch * 31) + threadID) = 1.000000e+00;
        *((real * )((char *) sv + pitch * 32) + threadID) = 1.000000e+00;
        *((real * )((char *) sv + pitch * 33) + threadID) = 2.424047e-01;
        *((real * )((char *) sv + pitch * 34) + threadID) = 1.795377e-04;
        *((real * )((char *) sv + pitch * 35) + threadID) = -6.883086e-25;
        *((real * )((char *) sv + pitch * 36) + threadID) = 1.117498e-02;
        *((real * )((char *) sv + pitch * 37) + threadID) = 9.980366e-01;
        *((real * )((char *) sv + pitch * 38) + threadID) = 8.588018e-04;
        *((real * )((char *) sv + pitch * 39) + threadID) = 7.097447e-04;
        *((real * )((char *) sv + pitch * 40) + threadID) = 3.812617e-04;
        *((real * )((char *) sv + pitch * 41) + threadID) = 1.357116e-05;
        *((real * )((char *) sv + pitch * 42) + threadID) = 2.302525e-23;
        *((real * )((char *) sv + pitch * 43) + threadID) = 1.561941e-04;
        *((real * )((char *) sv + pitch * 44) + threadID) = 2.351289e-04;
        *((real * )((char *) sv + pitch * 45) + threadID) = 8.077631e-03;
        *((real * )((char *) sv + pitch * 46) + threadID) = 9.993734e-01;
        *((real * )((char *) sv + pitch * 47) + threadID) = 0.000000e+00;
        *((real * )((char *) sv + pitch * 48) + threadID) = 0.000000e+00;
            
        if(use_adpt_dt) {
            *((real *)((char *)sv + pitch * 49) + threadID) = min_dt; // dt
            *((real *)((char *)sv + pitch * 50) + threadID) = 0.0;    // time_new
            *((real *)((char *)sv + pitch * 51) + threadID) = 0.0;    // previous dt
        }
    }
}

__global__ void kernel_set_model_initial_conditions_endo_mid_epi(real *sv, int num_volumes, size_t pitch, bool use_adpt_dt, real min_dt,\
                                                real *initial_endo, real *initial_epi, real *initial_mid, real *transmurality) {
    int threadID = blockDim.x * blockIdx.x + threadIdx.x;

    if (threadID < num_volumes) {
        for (int i = 0; i < NEQ; i++) {
            if (transmurality[threadID] == ENDO)
                *((real * )((char *) sv + pitch * i) + threadID) = initial_endo[i];
            else if (transmurality[threadID] == EPI)
                *((real * )((char *) sv + pitch * i) + threadID) = initial_epi[i];
            else
                *((real * )((char *) sv + pitch * i) + threadID) = initial_mid[i];
        }
            
        if(use_adpt_dt) {
            *((real *)((char *)sv + pitch * (NEQ+1)) + threadID) = min_dt; // dt
            *((real *)((char *)sv + pitch * (NEQ+2)) + threadID) = 0.0;    // time_new
            *((real *)((char *)sv + pitch * (NEQ+3)) + threadID) = 0.0;    // previous dt
        }
    }
}

extern "C" SET_ODE_INITIAL_CONDITIONS_GPU(set_model_initial_conditions_gpu) {

    size_t pitch_h;

    uint8_t use_adpt_dt = (uint8_t)solver->adaptive;

    log_info("Using GPU model implemented in %s\n", __FILE__);

    uint32_t num_volumes = solver->original_num_cells;

    if(use_adpt_dt) {
        log_info("Using Adaptive timestep to solve the ODEs\n");
    } else {
        log_info("Using Fixed timestep to solve the ODEs\n");
    }

    // execution configuration
    const int GRID = (num_volumes + BLOCK_SIZE - 1) / BLOCK_SIZE;

    size_t size = num_volumes * sizeof(real);

    if(use_adpt_dt)
        check_cuda_error(hipMallocPitch((void **)&(solver->sv), &pitch_h, size, (size_t)NEQ + 3));
    else
        check_cuda_error(hipMallocPitch((void **)&(solver->sv), &pitch_h, size, (size_t)NEQ));

    // Get initial condition from extra_data
    real *initial_conditions_endo = NULL;
    real *initial_conditions_epi = NULL;
    real *initial_conditions_mid = NULL;
    real *transmurality = NULL;
    real *initial_conditions_endo_device = NULL;
    real *initial_conditions_epi_device = NULL;
    real *initial_conditions_mid_device = NULL;
    real *transmurality_device = NULL;

    if(solver->ode_extra_data) {
        struct extra_data_for_torord_land *extra_data = (struct extra_data_for_torord_land*)solver->ode_extra_data;
        initial_conditions_endo = extra_data->initial_ss_endo;
        initial_conditions_epi = extra_data->initial_ss_epi;
        initial_conditions_mid = extra_data->initial_ss_mid;
        transmurality = extra_data->transmurality;
        check_cuda_error(hipMalloc((void **)&initial_conditions_endo_device, sizeof(real)*NEQ));
        check_cuda_error(hipMemcpy(initial_conditions_endo_device, initial_conditions_endo, sizeof(real)*NEQ, hipMemcpyHostToDevice));
        check_cuda_error(hipMalloc((void **)&initial_conditions_epi_device, sizeof(real)*NEQ));
        check_cuda_error(hipMemcpy(initial_conditions_epi_device, initial_conditions_epi, sizeof(real)*NEQ, hipMemcpyHostToDevice));
        check_cuda_error(hipMalloc((void **)&initial_conditions_mid_device, sizeof(real)*NEQ));
        check_cuda_error(hipMemcpy(initial_conditions_mid_device, initial_conditions_mid, sizeof(real)*NEQ, hipMemcpyHostToDevice));
        check_cuda_error(hipMalloc((void **)&transmurality_device, sizeof(real)*num_volumes));
        check_cuda_error(hipMemcpy(transmurality_device, transmurality, sizeof(real)*num_volumes, hipMemcpyHostToDevice));
    }
    else {
        log_info("[INFO] You should supply a mask function to tag the cells when using this mixed model!\n");
        log_info("[INFO] Considering all cells ENDO!\n");
    }

    if (solver->ode_extra_data) {
        kernel_set_model_initial_conditions_endo_mid_epi<<<GRID, BLOCK_SIZE>>>(solver->sv, num_volumes, pitch_h, use_adpt_dt, solver->min_dt,\
                                                            initial_conditions_endo_device, initial_conditions_epi_device, initial_conditions_mid_device,\
                                                            transmurality_device);
    }
    else {
        kernel_set_model_initial_conditions<<<GRID, BLOCK_SIZE>>>(solver->sv, num_volumes, pitch_h, use_adpt_dt, solver->min_dt);
    }
    
    check_cuda_error(hipPeekAtLastError());
    hipDeviceSynchronize();

    check_cuda_error(hipFree(initial_conditions_endo_device));
    check_cuda_error(hipFree(initial_conditions_epi_device));
    check_cuda_error(hipFree(initial_conditions_mid_device));
    check_cuda_error(hipFree(transmurality_device));

    return pitch_h;
}

extern "C" SOLVE_MODEL_ODES(solve_model_odes_gpu) {

    size_t num_cells_to_solve = ode_solver->num_cells_to_solve;
    uint32_t * cells_to_solve = ode_solver->cells_to_solve;
    real *sv = ode_solver->sv;
    real dt = ode_solver->min_dt;
    uint32_t num_steps = ode_solver->num_steps;

    // execution configuration
    const int GRID = ((int)num_cells_to_solve + BLOCK_SIZE - 1) / BLOCK_SIZE;

    size_t stim_currents_size = sizeof(real) * num_cells_to_solve;
    size_t cells_to_solve_size = sizeof(uint32_t) * num_cells_to_solve;

    real *stims_currents_device = NULL;
    check_cuda_error(hipMalloc((void **)&stims_currents_device, stim_currents_size));
    check_cuda_error(hipMemcpy(stims_currents_device, stim_currents, stim_currents_size, hipMemcpyHostToDevice));

    // the array cells to solve is passed when we are using and adaptive mesh
    uint32_t *cells_to_solve_device = NULL;
    if(cells_to_solve != NULL) {
        check_cuda_error(hipMalloc((void **)&cells_to_solve_device, cells_to_solve_size));
        check_cuda_error(hipMemcpy(cells_to_solve_device, cells_to_solve, cells_to_solve_size, hipMemcpyHostToDevice));
    }

    // Get the extra data array if exists
    uint32_t num_volumes = ode_solver->original_num_cells;
    real *transmurality = NULL;
    real *transmurality_device = NULL;
    int num_extra_parameters = 20;
    real extra_par[num_extra_parameters];
    real *extra_par_device = NULL;
    if(ode_solver->ode_extra_data) {
        struct extra_data_for_torord_land *extra_data = (struct extra_data_for_torord_land*)ode_solver->ode_extra_data;
        extra_par[0]  = extra_data->INa_Multiplier; 
        extra_par[1]  = extra_data->INaL_Multiplier;
        extra_par[2]  = extra_data->INaCa_Multiplier;
        extra_par[3]  = extra_data->INaK_Multiplier;
        extra_par[4]  = extra_data->INab_Multiplier; 
        extra_par[5]  = extra_data->Ito_Multiplier;
        extra_par[6]  = extra_data->IKr_Multiplier; 
        extra_par[7]  = extra_data->IKs_Multiplier; 
        extra_par[8]  = extra_data->IK1_Multiplier;
        extra_par[9]  = extra_data->IKb_Multiplier;
        extra_par[10]  = extra_data->IKCa_Multiplier;
        extra_par[11] = extra_data->ICaL_Multiplier;  
        extra_par[12] = extra_data->ICab_Multiplier;  
        extra_par[13] = extra_data->IpCa_Multiplier;
        extra_par[14] = extra_data->ICaCl_Multiplier; 
        extra_par[15] = extra_data->IClb_Multiplier;
        extra_par[16] = extra_data->Jrel_Multiplier;
        extra_par[17] = extra_data->Jup_Multiplier;
        extra_par[18] = extra_data->aCaMK_Multiplier;
        extra_par[19] = extra_data->taurelp_Multiplier;
        transmurality = extra_data->transmurality;
        
        check_cuda_error(hipMalloc((void **)&transmurality_device, sizeof(real)*num_volumes));
        check_cuda_error(hipMemcpy(transmurality_device, transmurality, sizeof(real)*num_volumes, hipMemcpyHostToDevice));

        check_cuda_error(hipMalloc((void **)&extra_par_device, sizeof(real)*num_extra_parameters));
        check_cuda_error(hipMemcpy(extra_par_device, extra_par, sizeof(real)*num_extra_parameters, hipMemcpyHostToDevice));
    }
    else {
        // Default: initialize all current modifiers
        for (uint32_t i = 0; i < num_extra_parameters; i++) {
            if (i == 10)
                extra_par[i] = 0.0;
            else 
                extra_par[i] = 1.0;
        }

        check_cuda_error(hipMalloc((void **)&extra_par_device, sizeof(real)*num_extra_parameters));
        check_cuda_error(hipMemcpy(extra_par_device, extra_par, sizeof(real)*num_extra_parameters, hipMemcpyHostToDevice));
    }

    // Transmurality mapping is defined on 'extra_data' function
    if (ode_solver->ode_extra_data) {
        solve_endo_mid_epi_gpu<<<GRID, BLOCK_SIZE>>>(current_t, dt, sv, stims_currents_device, cells_to_solve_device, transmurality_device, extra_par_device,\
                                    num_cells_to_solve, num_steps, ode_solver->pitch, ode_solver->adaptive, ode_solver->abs_tol, ode_solver->rel_tol, ode_solver->max_dt);
    }
    // No transmurality: all cells ENDO
    else {
        solve_gpu<<<GRID, BLOCK_SIZE>>>(current_t, dt, sv, stims_currents_device, cells_to_solve_device, extra_par_device,\
                                    num_cells_to_solve, num_steps, ode_solver->pitch, ode_solver->adaptive, ode_solver->abs_tol, ode_solver->rel_tol, ode_solver->max_dt);
    }

    check_cuda_error(hipPeekAtLastError());

    if (stims_currents_device) check_cuda_error(hipFree(stims_currents_device));
    if (cells_to_solve_device) check_cuda_error(hipFree(cells_to_solve_device));
    if (transmurality_device) check_cuda_error(hipFree(transmurality_device));
    if (extra_par_device) check_cuda_error(hipFree(extra_par_device));
}

__global__ void solve_gpu(real cur_time, real dt, real *sv, real *stim_currents, uint32_t *cells_to_solve, real *extra_params,\
                          uint32_t num_cells_to_solve, int num_steps, size_t pitch, bool use_adpt, real abstol, real reltol, real max_dt) {
    const real TOLERANCE = 1e-8;
    int threadID = blockDim.x * blockIdx.x + threadIdx.x;
    int sv_id;

    // Each thread solves one cell model
    if(threadID < num_cells_to_solve) {
        if(cells_to_solve)
            sv_id = cells_to_solve[threadID];
        else
            sv_id = threadID;

        if(!use_adpt) {
            real rDY[NEQ];
            real a[NEQ], b[NEQ];

            for(int n = 0; n < num_steps; ++n) {

                RHS_RL_gpu(a, b, sv, rDY, stim_currents[threadID], 0.0, extra_params, sv_id, dt, pitch, false);

                // Solve variables based on its type:
                //  Non-linear = Euler
                //  Hodkin-Huxley = Rush-Larsen || Euler (if 'a' coefficient is too small)
                SOLVE_EQUATION_EULER_GPU(0);        // v        
                SOLVE_EQUATION_EULER_GPU(1);        // nai    
                SOLVE_EQUATION_EULER_GPU(2);        // nass 
                SOLVE_EQUATION_EULER_GPU(3);        // ki 
                SOLVE_EQUATION_EULER_GPU(4);        // kss 
                SOLVE_EQUATION_EULER_GPU(5);        // cai   
                SOLVE_EQUATION_EULER_GPU(6);        // cass  
                SOLVE_EQUATION_EULER_GPU(7);        // cansr
                SOLVE_EQUATION_EULER_GPU(8);        // cajsr
                SOLVE_EQUATION_RUSH_LARSEN_GPU(9);  // m
                SOLVE_EQUATION_RUSH_LARSEN_GPU(10); // hp
                SOLVE_EQUATION_RUSH_LARSEN_GPU(11); // h
                SOLVE_EQUATION_RUSH_LARSEN_GPU(12); // j
                SOLVE_EQUATION_RUSH_LARSEN_GPU(13); // jp
                SOLVE_EQUATION_RUSH_LARSEN_GPU(14); // mL
                SOLVE_EQUATION_RUSH_LARSEN_GPU(15); // hL
                SOLVE_EQUATION_RUSH_LARSEN_GPU(16); // hLp
                SOLVE_EQUATION_RUSH_LARSEN_GPU(17); // a
                SOLVE_EQUATION_RUSH_LARSEN_GPU(18); // iF
                SOLVE_EQUATION_RUSH_LARSEN_GPU(19); // iS
                SOLVE_EQUATION_RUSH_LARSEN_GPU(20); // ap
                SOLVE_EQUATION_RUSH_LARSEN_GPU(21); // iFp
                SOLVE_EQUATION_RUSH_LARSEN_GPU(22); // iSp
                SOLVE_EQUATION_RUSH_LARSEN_GPU(23); // d
                SOLVE_EQUATION_RUSH_LARSEN_GPU(24); // ff
                SOLVE_EQUATION_RUSH_LARSEN_GPU(25); // fs
                SOLVE_EQUATION_RUSH_LARSEN_GPU(26); // fcaf
                SOLVE_EQUATION_RUSH_LARSEN_GPU(27); // fcas
                SOLVE_EQUATION_RUSH_LARSEN_GPU(28); // jca
                SOLVE_EQUATION_EULER_GPU(29);       // nca
                SOLVE_EQUATION_EULER_GPU(30);       // nca_i
                SOLVE_EQUATION_RUSH_LARSEN_GPU(31); // ffp
                SOLVE_EQUATION_RUSH_LARSEN_GPU(32); // fcafp
                SOLVE_EQUATION_RUSH_LARSEN_GPU(33); // xs1
                SOLVE_EQUATION_RUSH_LARSEN_GPU(34); // xs2
                SOLVE_EQUATION_RUSH_LARSEN_GPU(35); // Jrel_np
                SOLVE_EQUATION_EULER_GPU(36);       // CaMKt
                SOLVE_EQUATION_EULER_GPU(37);       // ikr_c0
                SOLVE_EQUATION_EULER_GPU(38);       // ikr_c1
                SOLVE_EQUATION_EULER_GPU(39);       // ikr_c2
                SOLVE_EQUATION_EULER_GPU(40);       // ikr_o
                SOLVE_EQUATION_EULER_GPU(41);       // ikr_i
                SOLVE_EQUATION_RUSH_LARSEN_GPU(42); // Jrel_p
                // ---------------------------------------------------
                // Land-Niederer
                SOLVE_EQUATION_EULER_GPU(43);       // XS
                SOLVE_EQUATION_EULER_GPU(44);       // XW
                SOLVE_EQUATION_EULER_GPU(45);       // Ca_TRPN
                SOLVE_EQUATION_EULER_GPU(46);       // TmBlocked
                SOLVE_EQUATION_EULER_GPU(47);       // ZETAS
                SOLVE_EQUATION_EULER_GPU(48);       // ZETAW
            }
        } else {
            solve_forward_euler_gpu_adpt(sv, stim_currents[threadID], 0.0, extra_params, cur_time + max_dt, sv_id, pitch, abstol,  reltol,  dt,  max_dt);
        }
    }
}

__global__ void solve_endo_mid_epi_gpu(real cur_time, real dt, real *sv, real *stim_currents, uint32_t *cells_to_solve, real *transmurality, real *extra_params,\
                          uint32_t num_cells_to_solve, int num_steps, size_t pitch, bool use_adpt, real abstol, real reltol, real max_dt) {
    const real TOLERANCE = 1e-8;
    int threadID = blockDim.x * blockIdx.x + threadIdx.x;
    int sv_id;

    // Each thread solves one cell model
    if(threadID < num_cells_to_solve) {
        if(cells_to_solve)
            sv_id = cells_to_solve[threadID];
        else
            sv_id = threadID;

        if(!use_adpt) {
            real rDY[NEQ];
            real a[NEQ], b[NEQ];

            for(int n = 0; n < num_steps; ++n) {

                RHS_RL_gpu(a, b, sv, rDY, stim_currents[threadID], transmurality[threadID], extra_params, sv_id, dt, pitch, false);

                // Solve variables based on its type:
                //  Non-linear = Euler
                //  Hodkin-Huxley = Rush-Larsen || Euler (if 'a' coefficient is too small)
                SOLVE_EQUATION_EULER_GPU(0);        // v        
                SOLVE_EQUATION_EULER_GPU(1);        // nai    
                SOLVE_EQUATION_EULER_GPU(2);        // nass 
                SOLVE_EQUATION_EULER_GPU(3);        // ki 
                SOLVE_EQUATION_EULER_GPU(4);        // kss 
                SOLVE_EQUATION_EULER_GPU(5);        // cai   
                SOLVE_EQUATION_EULER_GPU(6);        // cass  
                SOLVE_EQUATION_EULER_GPU(7);        // cansr
                SOLVE_EQUATION_EULER_GPU(8);        // cajsr
                SOLVE_EQUATION_RUSH_LARSEN_GPU(9);  // m
                SOLVE_EQUATION_RUSH_LARSEN_GPU(10); // hp
                SOLVE_EQUATION_RUSH_LARSEN_GPU(11); // h
                SOLVE_EQUATION_RUSH_LARSEN_GPU(12); // j
                SOLVE_EQUATION_RUSH_LARSEN_GPU(13); // jp
                SOLVE_EQUATION_RUSH_LARSEN_GPU(14); // mL
                SOLVE_EQUATION_RUSH_LARSEN_GPU(15); // hL
                SOLVE_EQUATION_RUSH_LARSEN_GPU(16); // hLp
                SOLVE_EQUATION_RUSH_LARSEN_GPU(17); // a
                SOLVE_EQUATION_RUSH_LARSEN_GPU(18); // iF
                SOLVE_EQUATION_RUSH_LARSEN_GPU(19); // iS
                SOLVE_EQUATION_RUSH_LARSEN_GPU(20); // ap
                SOLVE_EQUATION_RUSH_LARSEN_GPU(21); // iFp
                SOLVE_EQUATION_RUSH_LARSEN_GPU(22); // iSp
                SOLVE_EQUATION_RUSH_LARSEN_GPU(23); // d
                SOLVE_EQUATION_RUSH_LARSEN_GPU(24); // ff
                SOLVE_EQUATION_RUSH_LARSEN_GPU(25); // fs
                SOLVE_EQUATION_RUSH_LARSEN_GPU(26); // fcaf
                SOLVE_EQUATION_RUSH_LARSEN_GPU(27); // fcas
                SOLVE_EQUATION_RUSH_LARSEN_GPU(28); // jca
                SOLVE_EQUATION_EULER_GPU(29);       // nca
                SOLVE_EQUATION_EULER_GPU(30);       // nca_i
                SOLVE_EQUATION_RUSH_LARSEN_GPU(31); // ffp
                SOLVE_EQUATION_RUSH_LARSEN_GPU(32); // fcafp
                SOLVE_EQUATION_RUSH_LARSEN_GPU(33); // xs1
                SOLVE_EQUATION_RUSH_LARSEN_GPU(34); // xs2
                SOLVE_EQUATION_RUSH_LARSEN_GPU(35); // Jrel_np
                SOLVE_EQUATION_EULER_GPU(36);       // CaMKt
                SOLVE_EQUATION_EULER_GPU(37);       // ikr_c0
                SOLVE_EQUATION_EULER_GPU(38);       // ikr_c1
                SOLVE_EQUATION_EULER_GPU(39);       // ikr_c2
                SOLVE_EQUATION_EULER_GPU(40);       // ikr_o
                SOLVE_EQUATION_EULER_GPU(41);       // ikr_i
                SOLVE_EQUATION_RUSH_LARSEN_GPU(42); // Jrel_p
                // ---------------------------------------------------
                // Land-Niederer
                SOLVE_EQUATION_EULER_GPU(43);       // XS
                SOLVE_EQUATION_EULER_GPU(44);       // XW
                SOLVE_EQUATION_EULER_GPU(45);       // Ca_TRPN
                SOLVE_EQUATION_EULER_GPU(46);       // TmBlocked
                SOLVE_EQUATION_EULER_GPU(47);       // ZETAS
                SOLVE_EQUATION_EULER_GPU(48);       // ZETAW
            }
        } else {
            solve_forward_euler_gpu_adpt(sv, stim_currents[threadID], transmurality[threadID], extra_params, cur_time + max_dt, sv_id, pitch, abstol,  reltol,  dt,  max_dt);
        }
    }
}

inline __device__ void solve_forward_euler_gpu_adpt(real *sv, real stim_curr, real mapping, real *extra_params, real final_time, int thread_id, size_t pitch, real abstol, real reltol, real min_dt, real max_dt) {

    #define DT *((real *)((char *)sv + pitch * (NEQ)) + thread_id)
    #define TIME_NEW *((real *)((char *)sv + pitch * (NEQ+1)) + thread_id)
    #define PREVIOUS_DT *((real *)((char *)sv + pitch * (NEQ+2)) + thread_id)

    real rDY[NEQ];

    real _tolerances_[NEQ];
    real _aux_tol = 0.0;
    real dt = DT;
    real time_new = TIME_NEW;
    real previous_dt = PREVIOUS_DT;

    real edos_old_aux_[NEQ];
    real edos_new_euler_[NEQ];
    real _k1__[NEQ];
    real _k2__[NEQ];
    real _k_aux__[NEQ];
    real sv_local[NEQ];

    const real _beta_safety_ = 0.8;

    const real __tiny_ = pow(abstol, 2.0);

    if(time_new + dt > final_time) {
        dt = final_time - time_new;
    }

    for(int i = 0; i < NEQ; i++) {
        sv_local[i] = *((real *)((char *)sv + pitch * i) + thread_id);
    }

    RHS_gpu(sv_local, rDY, stim_curr, mapping, extra_params, thread_id, dt, pitch, true);
    time_new += dt;

    for(int i = 0; i < NEQ; i++) {
        _k1__[i] = rDY[i];
    }

	while(1) {

		for(int i = 0; i < NEQ; i++) {
			// stores the old variables in a vector
			edos_old_aux_[i] = sv_local[i];
			// computes euler method
			edos_new_euler_[i] = _k1__[i] * dt + edos_old_aux_[i];
			// steps ahead to compute the rk2 method
			sv_local[i] = edos_new_euler_[i];
		}

		time_new += dt;

		RHS_gpu(sv_local, rDY, stim_curr, mapping, extra_params, thread_id, dt, pitch, true);
		time_new -= dt; // step back

		real greatestError = 0.0, auxError = 0.0;
		
		for(int i = 0; i < NEQ; i++) {

			// stores the new evaluation
			_k2__[i] = rDY[i];
			_aux_tol = fabs(edos_new_euler_[i]) * reltol;
			_tolerances_[i] = (abstol > _aux_tol) ? abstol : _aux_tol;

			// finds the greatest error between  the steps
			auxError = fabs(((dt / 2.0) * (_k1__[i] - _k2__[i])) / _tolerances_[i]);

			greatestError = (auxError > greatestError) ? auxError : greatestError;
		}

		/// adapt the time step
		greatestError += __tiny_;
		previous_dt = dt;

		/// adapt the time step
		dt = _beta_safety_ * dt * sqrt(1.0f / greatestError);

		if(dt < min_dt) {
			dt = min_dt;
		}
		else if(dt > max_dt) {
			dt = max_dt;
		}

		if(time_new + dt > final_time) {
			dt = final_time - time_new;
		}

		// it doesn't accept the solution or accept and risk a NaN
		if(greatestError >= 1.0f && dt > min_dt) {
			// restore the old values to do it again
			for(int i = 0; i < NEQ; i++) {
				sv_local[i] = edos_old_aux_[i];
			}
		
		} else {
			for(int i = 0; i < NEQ; i++) {
				_k_aux__[i] = _k2__[i];
				_k2__[i] = _k1__[i];
				_k1__[i] = _k_aux__[i];
			}

			for(int i = 0; i < NEQ; i++) {
				sv_local[i] = edos_new_euler_[i];
			}

			if(time_new + previous_dt >= final_time) {
				if(final_time == time_new) {
					break;
				} else if(time_new < final_time) {
					dt = previous_dt = final_time - time_new;
					time_new += previous_dt;
					break;
				} 	
			} else {
				time_new += previous_dt;
			}
		}
	}

    for(int i = 0; i < NEQ; i++) {
        *((real *)((char *)sv + pitch * i) + thread_id) = sv_local[i];
    }

    DT = dt;
    TIME_NEW = time_new;
    PREVIOUS_DT = previous_dt;
}

inline __device__ void RHS_gpu(real *sv, real *rDY_, real stim_current, \
                            real mapping, real *extra_params, int threadID_, real dt, size_t pitch, bool use_adpt_dt) {
    
    // Current modifiers
    real INa_Multiplier = extra_params[0];   
    real INaL_Multiplier = extra_params[1];  
    real INaCa_Multiplier = extra_params[2];  
    real INaK_Multiplier = extra_params[3];  
    real INab_Multiplier = extra_params[4];   
    real Ito_Multiplier = extra_params[5];  
    real IKr_Multiplier = extra_params[6];   
    real IKs_Multiplier = extra_params[7];   
    real IK1_Multiplier = extra_params[8];  
    real IKb_Multiplier = extra_params[9];  
    real IKCa_Multiplier = extra_params[10];  
    real ICaL_Multiplier = extra_params[11];   
    real ICab_Multiplier = extra_params[12];   
    real IpCa_Multiplier = extra_params[13]; 
    real ICaCl_Multiplier = extra_params[14];  
    real IClb_Multiplier = extra_params[15]; 
    real Jrel_Multiplier = extra_params[16]; 
    real Jup_Multiplier = extra_params[17]; 
    real aCaMK_Multiplier = extra_params[18]; 
    real taurelp_Multiplier = extra_params[19];

    // Get the celltype for the current cell
    real celltype = mapping;
    
    // Get the stimulus current from the current cell
    real calc_I_stim = stim_current;

    // Constant variables
    const real cli = 24;   // Intracellular Cl  [mM]
    const real clo = 150;  // Extracellular Cl  [mM]

    // State variables
    real v;
    real nai;
    real nass;
    real ki;
    real kss;
    real cai;
    real cass;
    real cansr;
    real cajsr;
    real m;
    real hp;
    real h;
    real j;
    real jp;
    real mL;
    real hL;
    real hLp;
    real a;
    real iF;
    real iS;
    real ap;
    real iFp;
    real iSp;

    // ical
    real d;
    real ff;
    real fs;
    real fcaf;
    real fcas;
    real jca;
    real nca;
    real nca_i;
    real ffp;
    real fcafp;

    real xs1;
    real xs2;
    real Jrel_np;
    real CaMKt;

    // new MM ICaL states
    real ikr_c0;
    real ikr_c1;
    real ikr_c2;
    real ikr_o;
    real ikr_i;
    real Jrel_p;

    real XS;
    real XW;
    real Ca_TRPN;
    real TmBlocked;
    real ZETAS;
    real ZETAW;

    real TA;

    v = *((real *)((char *)sv + pitch * 0) + threadID_);
    nai = *((real *)((char *)sv + pitch * 1) + threadID_);
    nass = *((real *)((char *)sv + pitch * 2) + threadID_);
    ki = *((real *)((char *)sv + pitch * 3) + threadID_);
    kss = *((real *)((char *)sv + pitch * 4) + threadID_);
    cai = *((real *)((char *)sv + pitch * 5) + threadID_);
    cass = *((real *)((char *)sv + pitch * 6) + threadID_);
    cansr = *((real *)((char *)sv + pitch * 7) + threadID_);
    cajsr = *((real *)((char *)sv + pitch * 8) + threadID_);
    m = *((real *)((char *)sv + pitch * 9) + threadID_);
    hp = *((real *)((char *)sv + pitch * 10) + threadID_);
    h = *((real *)((char *)sv + pitch * 11) + threadID_);
    j = *((real *)((char *)sv + pitch * 12) + threadID_);
    jp = *((real *)((char *)sv + pitch * 13) + threadID_);
    mL = *((real *)((char *)sv + pitch * 14) + threadID_);
    hL = *((real *)((char *)sv + pitch * 15) + threadID_);
    hLp = *((real *)((char *)sv + pitch * 16) + threadID_);
    a = *((real *)((char *)sv + pitch * 17) + threadID_);
    iF = *((real *)((char *)sv + pitch * 18) + threadID_);
    iS = *((real *)((char *)sv + pitch * 19) + threadID_);
    ap = *((real *)((char *)sv + pitch * 20) + threadID_);
    iFp = *((real *)((char *)sv + pitch * 21) + threadID_);
    iSp = *((real *)((char *)sv + pitch * 22) + threadID_);

    // ical
    d = *((real *)((char *)sv + pitch * 23) + threadID_);
    ff = *((real *)((char *)sv + pitch * 24) + threadID_);
    fs = *((real *)((char *)sv + pitch * 25) + threadID_);
    fcaf = *((real *)((char *)sv + pitch * 26) + threadID_);
    fcas = *((real *)((char *)sv + pitch * 27) + threadID_);
    jca = *((real *)((char *)sv + pitch * 28) + threadID_);
    nca = *((real *)((char *)sv + pitch * 29) + threadID_);
    nca_i = *((real *)((char *)sv + pitch * 30) + threadID_);
    ffp = *((real *)((char *)sv + pitch * 31) + threadID_);
    fcafp = *((real *)((char *)sv + pitch * 32) + threadID_);

    xs1 = *((real *)((char *)sv + pitch * 33) + threadID_);
    xs2 = *((real *)((char *)sv + pitch * 34) + threadID_);
    Jrel_np = *((real *)((char *)sv + pitch * 35) + threadID_);
    CaMKt = *((real *)((char *)sv + pitch * 36) + threadID_);

    // new MM ICaL states
    ikr_c0 = *((real *)((char *)sv + pitch * 37) + threadID_);
    ikr_c1 = *((real *)((char *)sv + pitch * 38) + threadID_);
    ikr_c2 = *((real *)((char *)sv + pitch * 39) + threadID_);
    ikr_o = *((real *)((char *)sv + pitch * 40) + threadID_);
    ikr_i = *((real *)((char *)sv + pitch * 41) + threadID_);
    Jrel_p = *((real *)((char *)sv + pitch * 42) + threadID_);

    // Land-Niederer model
    XS = fmaxf(0,(*((real *)((char *)sv + pitch * 43) + threadID_)));
    XW = fmaxf(0,(*((real *)((char *)sv + pitch * 44) + threadID_)));
    Ca_TRPN = fmaxf(0,(*((real *)((char *)sv + pitch * 45) + threadID_)));
    TmBlocked = (*((real *)((char *)sv + pitch * 46) + threadID_));
    ZETAS = (*((real *)((char *)sv + pitch * 47) + threadID_));
    ZETAW = (*((real *)((char *)sv + pitch * 48) + threadID_));

    #include "ToRORd_Land_mixed_endo_mid_epi.common.c"
}

inline __device__ void RHS_RL_gpu(real *a_, real *b_, real *sv, real *rDY_, real stim_current, \
                                real mapping, real *extra_params, int threadID_, real dt, size_t pitch, bool use_adpt_dt) {
    
    // Current modifiers
    real INa_Multiplier = extra_params[0];
    real INaL_Multiplier = extra_params[1];  
    real INaCa_Multiplier = extra_params[2];  
    real INaK_Multiplier = extra_params[3];  
    real INab_Multiplier = extra_params[4];   
    real Ito_Multiplier = extra_params[5];  
    real IKr_Multiplier = extra_params[6];   
    real IKs_Multiplier = extra_params[7];   
    real IK1_Multiplier = extra_params[8];  
    real IKb_Multiplier = extra_params[9];  
    real IKCa_Multiplier = extra_params[10];  
    real ICaL_Multiplier = extra_params[11];   
    real ICab_Multiplier = extra_params[12];   
    real IpCa_Multiplier = extra_params[13]; 
    real ICaCl_Multiplier = extra_params[14];  
    real IClb_Multiplier = extra_params[15]; 
    real Jrel_Multiplier = extra_params[16]; 
    real Jup_Multiplier = extra_params[17]; 
    real aCaMK_Multiplier = extra_params[18]; 
    real taurelp_Multiplier = extra_params[19];

    // Get the celltype for the current cell
    real celltype = mapping;
    
    // Get the stimulus current from the current cell
    real calc_I_stim = stim_current;

    // Constant variables
    const real cli = 24;   // Intracellular Cl  [mM]
    const real clo = 150;  // Extracellular Cl  [mM]

    // State variables
    real v;
    real nai;
    real nass;
    real ki;
    real kss;
    real cai;
    real cass;
    real cansr;
    real cajsr;
    real m;
    real hp;
    real h;
    real j;
    real jp;
    real mL;
    real hL;
    real hLp;
    real a;
    real iF;
    real iS;
    real ap;
    real iFp;
    real iSp;

    // ical
    real d;
    real ff;
    real fs;
    real fcaf;
    real fcas;
    real jca;
    real nca;
    real nca_i;
    real ffp;
    real fcafp;

    real xs1;
    real xs2;
    real Jrel_np;
    real CaMKt;

    // new MM ICaL states
    real ikr_c0;
    real ikr_c1;
    real ikr_c2;
    real ikr_o;
    real ikr_i;
    real Jrel_p;

    real XS;
    real XW;
    real Ca_TRPN;
    real TmBlocked;
    real ZETAS;
    real ZETAW;

    real TA;

    v = *((real *)((char *)sv + pitch * 0) + threadID_);
    nai = *((real *)((char *)sv + pitch * 1) + threadID_);
    nass = *((real *)((char *)sv + pitch * 2) + threadID_);
    ki = *((real *)((char *)sv + pitch * 3) + threadID_);
    kss = *((real *)((char *)sv + pitch * 4) + threadID_);
    cai = *((real *)((char *)sv + pitch * 5) + threadID_);
    cass = *((real *)((char *)sv + pitch * 6) + threadID_);
    cansr = *((real *)((char *)sv + pitch * 7) + threadID_);
    cajsr = *((real *)((char *)sv + pitch * 8) + threadID_);
    m = *((real *)((char *)sv + pitch * 9) + threadID_);
    hp = *((real *)((char *)sv + pitch * 10) + threadID_);
    h = *((real *)((char *)sv + pitch * 11) + threadID_);
    j = *((real *)((char *)sv + pitch * 12) + threadID_);
    jp = *((real *)((char *)sv + pitch * 13) + threadID_);
    mL = *((real *)((char *)sv + pitch * 14) + threadID_);
    hL = *((real *)((char *)sv + pitch * 15) + threadID_);
    hLp = *((real *)((char *)sv + pitch * 16) + threadID_);
    a = *((real *)((char *)sv + pitch * 17) + threadID_);
    iF = *((real *)((char *)sv + pitch * 18) + threadID_);
    iS = *((real *)((char *)sv + pitch * 19) + threadID_);
    ap = *((real *)((char *)sv + pitch * 20) + threadID_);
    iFp = *((real *)((char *)sv + pitch * 21) + threadID_);
    iSp = *((real *)((char *)sv + pitch * 22) + threadID_);

    // ical
    d = *((real *)((char *)sv + pitch * 23) + threadID_);
    ff = *((real *)((char *)sv + pitch * 24) + threadID_);
    fs = *((real *)((char *)sv + pitch * 25) + threadID_);
    fcaf = *((real *)((char *)sv + pitch * 26) + threadID_);
    fcas = *((real *)((char *)sv + pitch * 27) + threadID_);
    jca = *((real *)((char *)sv + pitch * 28) + threadID_);
    nca = *((real *)((char *)sv + pitch * 29) + threadID_);
    nca_i = *((real *)((char *)sv + pitch * 30) + threadID_);
    ffp = *((real *)((char *)sv + pitch * 31) + threadID_);
    fcafp = *((real *)((char *)sv + pitch * 32) + threadID_);

    xs1 = *((real *)((char *)sv + pitch * 33) + threadID_);
    xs2 = *((real *)((char *)sv + pitch * 34) + threadID_);
    Jrel_np = *((real *)((char *)sv + pitch * 35) + threadID_);
    CaMKt = *((real *)((char *)sv + pitch * 36) + threadID_);

    // new MM ICaL states
    ikr_c0 = *((real *)((char *)sv + pitch * 37) + threadID_);
    ikr_c1 = *((real *)((char *)sv + pitch * 38) + threadID_);
    ikr_c2 = *((real *)((char *)sv + pitch * 39) + threadID_);
    ikr_o = *((real *)((char *)sv + pitch * 40) + threadID_);
    ikr_i = *((real *)((char *)sv + pitch * 41) + threadID_);
    Jrel_p = *((real *)((char *)sv + pitch * 42) + threadID_);

    // Land-Niederer model
    XS = fmaxf(0,(*((real *)((char *)sv + pitch * 43) + threadID_)));
    XW = fmaxf(0,(*((real *)((char *)sv + pitch * 44) + threadID_)));
    Ca_TRPN = fmaxf(0,(*((real *)((char *)sv + pitch * 45) + threadID_)));
    TmBlocked = (*((real *)((char *)sv + pitch * 46) + threadID_));
    ZETAS = (*((real *)((char *)sv + pitch * 47) + threadID_));
    ZETAW = (*((real *)((char *)sv + pitch * 48) + threadID_));

    #include "ToRORd_Land_mixed_endo_mid_epi_RL.common.c"
}
